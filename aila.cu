#include "hip/hip_runtime.h"
// CudaTracerKernels.hpp
#pragma once
#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"

//------------------------------------------------------------------------
// Constants.
//------------------------------------------------------------------------

enum
{
    MaxBlockHeight      = 6,            // Upper bound for blockDim.y.
    EntrypointSentinel  = 0x76543210,   // Bottom-most stack entry, indicating the end of traversal.
};

#define TRACE_FUNC \
    extern "C" __global__ void trace( \
        int             numRays,        /* Total number of rays in the batch. */ \
        bool            anyHit,         /* False if rays need to find the closest hit. */ \
        float4*         rays,           /* Ray input: float3 origin, float tmin, float3 direction, float tmax. */ \
        int4*           results,        /* Ray output: int triangleID, float hitT, int2 padding. */ \
        float4*         nodesA,         /* SOA: bytes 0-15 of each node, AOS/Compact: 64 bytes per node. */ \
        float4*         nodesB,         /* SOA: bytes 16-31 of each node, AOS/Compact: unused. */ \
        float4*         nodesC,         /* SOA: bytes 32-47 of each node, AOS/Compact: unused. */ \
        float4*         nodesD,         /* SOA: bytes 48-63 of each node, AOS/Compact: unused. */ \
        float4*         trisA,          /* SOA: bytes 0-15 of each triangle, AOS: 64 bytes per triangle, Compact: 48 bytes per triangle. */ \
        float4*         trisB,          /* SOA: bytes 16-31 of each triangle, AOS/Compact: unused. */ \
        float4*         trisC,          /* SOA: bytes 32-47 of each triangle, AOS/Compact: unused. */ \
        int*            triIndices)     /* Triangle index remapping table. */

struct RayStruct
{
    float   idirx;  // 1.0f / ray.direction.x
    float   idiry;  // 1.0f / ray.direction.y
    float   idirz;  // 1.0f / ray.direction.z
    float   tmin;   // ray.tminq
    float   dummy;  // Padding to avoid bank conflicts.
};

//------------------------------------------------------------------------
// Utilities.
//------------------------------------------------------------------------

#define FETCH_GLOBAL(NAME, IDX, TYPE) ((const TYPE*)NAME)[IDX]
#define FETCH_TEXTURE(NAME, IDX, TYPE) tex1Dfetch(t_ ## NAME, IDX)
#define FETCH_ARRAY(NAME, IDX, TYPE) NAME[IDX]
#define STORE_RESULT(RAY, TRI, T, U, V) ((int4*)results)[(RAY)] = make_int4(TRI, __float_as_int(T), __float_as_int(U), __float_as_int(V))

//------------------------------------------------------------------------

#ifdef __HIPCC__

template <class T> __device__ __inline__ void swap(T& a,T& b)
{
    T t = a;
    a = b;
    b = t;
}

// Using video instructions
__device__ __inline__ int   min_min   (int a, int b, int c) { int v; asm("vmin.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   min_max   (int a, int b, int c) { int v; asm("vmin.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   max_min   (int a, int b, int c) { int v; asm("vmax.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   max_max   (int a, int b, int c) { int v; asm("vmax.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }

__device__ __inline__ float fmin_fmin (float a, float b, float c) { return __int_as_float(min_min(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmin_fmax (float a, float b, float c) { return __int_as_float(min_max(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmin (float a, float b, float c) { return __int_as_float(max_min(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmax (float a, float b, float c) { return __int_as_float(max_max(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }

// Experimentally determined best mix of float/int/video minmax instructions for Kepler.
__device__ __inline__ float spanBeginKepler(float a0, float a1, float b0, float b1, float c0, float c1, float d) {	return fmax_fmax( fminf(a0,a1), fminf(b0,b1), fmin_fmax(c0, c1, d)); }
__device__ __inline__ float spanEndKepler(float a0, float a1, float b0, float b1, float c0, float c1, float d)	{	return fmin_fmin( fmaxf(a0,a1), fmaxf(b0,b1), fmax_fmin(c0, c1, d)); }

#endif

//------------------------------------------------------------------------
// kepler_dynamic_fetch.cu

#include "CudaTracerKernels.hpp"

#define STACK_SIZE              64          // Size of the traversal stack in local memory.
#define DYNAMIC_FETCH_THRESHOLD 20          // If fewer than this active, fetch new rays


extern "C" __device__ int g_warpCounter;    // Work counter for persistent threads.

TRACE_FUNC
{
    // Traversal stack in CUDA thread-local memory.

    int traversalStack[STACK_SIZE];
    traversalStack[0] = EntrypointSentinel; // Bottom-most entry.

    // Live state during traversal, stored in registers.

    float   origx, origy, origz;            // Ray origin.
    char*   stackPtr;                       // Current position in traversal stack.
    int     leafAddr;                       // First postponed leaf, non-negative if none.
    //int     leafAddr2;                      // Second postponed leaf, non-negative if none.
    int     nodeAddr = EntrypointSentinel;  // Non-negative: current internal node, negative: second postponed leaf.
    int     hitIndex;                       // Triangle index of the closest intersection, -1 if none.
    float   hitT;                           // t-value of the closest intersection.
    float   hitU;
    float   hitV;
    float   tmin;
    int     rayidx;
    float   oodx;
    float   oody;
    float   oodz;
    float   dirx;
    float   diry;
    float   dirz;
    float   idirx;
    float   idiry;
    float   idirz;

	
    // Initialize persistent threads.

    __shared__ volatile int nextRayArray[MaxBlockHeight]; // Current ray index in global buffer.
    // Persistent threads: fetch and process rays in a loop.

    do
    {
        const int tidx = threadIdx.x;
        volatile int& rayBase = nextRayArray[threadIdx.y];

        // Fetch new rays from the global pool using lane 0.

        const bool          terminated     = nodeAddr==EntrypointSentinel;
        const unsigned int  maskTerminated = __ballot(terminated);
        const int           numTerminated  = __popc(maskTerminated);
        const int           idxTerminated  = __popc(maskTerminated & ((1u<<tidx)-1));
    
        if(terminated)
        {
            if (idxTerminated == 0)
                rayBase = atomicAdd(&g_warpCounter, numTerminated);

            rayidx = rayBase + idxTerminated;
            if (rayidx >= numRays)
                break;

            // Fetch ray.

            float4 o = FETCH_GLOBAL(rays, rayidx * 2 + 0, float4);
            float4 d = FETCH_GLOBAL(rays, rayidx * 2 + 1, float4);
            origx = o.x;
            origy = o.y;
            origz = o.z;
            tmin  = o.w;
            dirx  = d.x;
            diry  = d.y;
            dirz  = d.z;
            hitT  = d.w;

             float ooeps = exp2f(-80.0f); // Avoid div by zero.
            idirx = 1.0f / (fabsf(d.x) > ooeps ? d.x : copysignf(ooeps, d.x));
            idiry = 1.0f / (fabsf(d.y) > ooeps ? d.y : copysignf(ooeps, d.y));
            idirz = 1.0f / (fabsf(d.z) > ooeps ? d.z : copysignf(ooeps, d.z));
            oodx  = origx * idirx;
            oody  = origy * idiry;
            oodz  = origz * idirz;

            // Setup traversal.

            stackPtr = (char*)&traversalStack[0];
            leafAddr = 0;   // No postponed leaf.
            //leafAddr2= 0;   // No postponed leaf.
            nodeAddr = 0;   // Start from the root.
            hitIndex = -1;  // No triangle intersected so far.

        }

        // Traversal loop.

        while(nodeAddr != EntrypointSentinel)
        {
            // Traverse internal nodes until all SIMD lanes have found a leaf.

//          while (nodeAddr >= 0 && nodeAddr != EntrypointSentinel)
            while ((unsigned int)(nodeAddr) < (unsigned int)(EntrypointSentinel))   // functionally equivalent, but faster
            {
                // Fetch AABBs of the two child nodes.

                const float4 n0xy = tex1Dfetch(t_nodesA, nodeAddr + 0); // (c0.lo.x, c0.hi.x, c0.lo.y, c0.hi.y)
                const float4 n1xy = tex1Dfetch(t_nodesA, nodeAddr + 1); // (c1.lo.x, c1.hi.x, c1.lo.y, c1.hi.y)
                const float4 nz   = tex1Dfetch(t_nodesA, nodeAddr + 2); // (c0.lo.z, c0.hi.z, c1.lo.z, c1.hi.z)
                      float4 tmp  = tex1Dfetch(t_nodesA, nodeAddr + 3); // child_index0, child_index1

                int2  cnodes= *(int2*)&tmp;

                // Intersect the ray against the child nodes.

                const float c0lox = n0xy.x * idirx - oodx;
                const float c0hix = n0xy.y * idirx - oodx;
                const float c0loy = n0xy.z * idiry - oody;
                const float c0hiy = n0xy.w * idiry - oody;
                const float c0loz = nz.x   * idirz - oodz;
                const float c0hiz = nz.y   * idirz - oodz;
                const float c1loz = nz.z   * idirz - oodz;
                const float c1hiz = nz.w   * idirz - oodz;
                const float c0min = spanBeginKepler(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, tmin);
                const float c0max = spanEndKepler  (c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, hitT);
                const float c1lox = n1xy.x * idirx - oodx;
                const float c1hix = n1xy.y * idirx - oodx;
                const float c1loy = n1xy.z * idiry - oody;
                const float c1hiy = n1xy.w * idiry - oody;
                const float c1min = spanBeginKepler(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, tmin);
                const float c1max = spanEndKepler  (c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, hitT);

                bool swp = (c1min < c0min);

                bool traverseChild0 = (c0max >= c0min);
                bool traverseChild1 = (c1max >= c1min);

                // Neither child was intersected => pop stack.

                if (!traverseChild0 && !traverseChild1)
                {
                    nodeAddr = *(int*)stackPtr;
                    stackPtr -= 4;
                }

                // Otherwise => fetch child pointers.

                else
                {
                    nodeAddr = (traverseChild0) ? cnodes.x : cnodes.y;

                    // Both children were intersected => push the farther one.

                    if (traverseChild0 && traverseChild1)
                    {
                        if (swp)
                            swap(nodeAddr, cnodes.y);
                        stackPtr += 4;
                        *(int*)stackPtr = cnodes.y;
                    }
                }

                // First leaf => postpone and continue traversal.

                if (nodeAddr < 0 && leafAddr  >= 0)     // Postpone max 1
//              if (nodeAddr < 0 && leafAddr2 >= 0)     // Postpone max 2
                {
                    //leafAddr2= leafAddr;          // postpone 2
                    leafAddr = nodeAddr;
                    nodeAddr = *(int*)stackPtr;
                    stackPtr -= 4;
                }

                // All SIMD lanes have found a leaf? => process them.

                // NOTE: inline PTX implementation of "if(!__any(leafAddr >= 0)) break;".
                // tried everything with CUDA 4.2 but always got several redundant instructions.

                unsigned int mask;
                asm("{\n"
                    "   .reg .pred p;               \n"
                    "setp.ge.s32        p, %1, 0;   \n"
                    "vote.ballot.b32    %0,p;       \n"
                    "}"
                    : "=r"(mask)
                    : "r"(leafAddr));
                if(!mask)
                    break;

                //if(!__any(leafAddr >= 0))
                //    break;
            }

            // Process postponed leaf nodes.

            while (leafAddr < 0)
            {
            
            
            	
			    for (int triAddr = ~leafAddr;; triAddr += 3)
                {
                	
                	 // Tris in TEX (good to fetch as a single batch)
                    const float3 v00 = make_float3(tex1Dfetch(t_trisA, triAddr + 0));
                    const float3 v11 = make_float3(tex1Dfetch(t_trisA, triAddr + 1));
                    const float3 v22 = make_float3(tex1Dfetch(t_trisA, triAddr + 2));

                    // ------ Modified  version of the intersection routine -----------

                    // End marker (negative zero) => all triangles processed.
                    if (__float_as_int(v00.x) == 0x80000000)
                        break;

					const float3 e1 = v00 - v11;
                    const float3 e2 = v22 - v00;
                    const float3 n = cross(e1, e2);
                    const float3 dir = make_float3(dirx,diry,dirz);
                    const float3 org = make_float3(origx, origy, origz);
                    const float3 c = v00 - org;
                    const float3 r = cross(dir, c);
                    const float det = dot(n, dir);
                    const float abs_det = fabsf(det);

                    const float u = __int_as_float(__float_as_int(dot(r, e2)) ^ (__float_as_int(det) & 0x80000000));
                    bool mask = u >= 0.0f;
                    const float v = __int_as_float(__float_as_int(dot(r, e1)) ^ (__float_as_int(det) & 0x80000000));
                    mask &= v >= 0.0f;
                    const float w = abs_det - u - v;
                    mask &= w >= 0.0f;

                    if (mask) {
                        const float t = __int_as_float(__float_as_int(dot(n, c)) ^ (__float_as_int(det) & 0x80000000));
                        mask &= (t >= abs_det * tmin) & (t <= abs_det * hitT) & (det != 0.0f);
                        if (mask) {
                            const float inv_det = 1.0f / abs_det;
                            hitT = t * inv_det;
                            hitU = u * inv_det;
                            hitV = v * inv_det;
                            hitIndex = triAddr;
                            if (anyHit) {
                                nodeAddr = EntrypointSentinel;
                                break;
                            }
                        }
                    }
				
                } // triangle

                // Another leaf was postponed => process it as well.

//              if(leafAddr2<0) { leafAddr = leafAddr2; leafAddr2=0; } else     // postpone2
                {
                    leafAddr = nodeAddr;
                    if (nodeAddr < 0)
                    {
                        nodeAddr = *(int*)stackPtr;
                        stackPtr -= 4;
                    }
                }
            } // leaf

            // DYNAMIC FETCH

            if( __popc(__ballot(true)) < DYNAMIC_FETCH_THRESHOLD )
                break;

        } // traversal

        // Remap intersected triangle index, and store the result.

        if (hitIndex == -1) { STORE_RESULT(rayidx, -1, hitT, hitU, hitV); }
        else                { STORE_RESULT(rayidx, FETCH_TEXTURE(triIndices, hitIndex, int), hitT, hitU, hitV); }

    } while(true);
}

int checkCudaError() {
    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        printf("CUDA error: %s\n", hipGetErrorString(error));
		return 1;
    }
    return 0;
}

int getFuncSharedSize() {
    hipFuncAttributes  	attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>("trace"));
    checkCudaError();
    return attr.sharedSizeBytes;
}

void resetKernel() {
    //Reset Warp Counter
    int reset=0;
    hipMemcpyToSymbol(HIP_SYMBOL(g_warpCounter), &reset, sizeof(int), 0, hipMemcpyHostToDevice);
    checkCudaError();
}

void launchKernel(int2 			  gridSize,    \
				  int2 			  blockSize,   \
				  int             numRays,     \
                  int             numNodes,    \
                  int             numTris,     \
		          float4*         rays_h,      \
		          float4*         rays_d,      \
    			  int4*           results_h,   \
    			  int4*           results_d,   \
    			  float4*         nodesA_h,    \
    			  float4*         nodesA_d,    \
				  float4*         trisA_h,     \
				  float4*         trisA_d,     \
				  int*            triIndices_h \
                  int*            triIndices_d) {
    // Kernel invocation
    dim3 numBlocks(gridSize.x, gridSize.y);
    dim3 threadsPerBlock(blockSize.x, blockSize.y);

#define RAY_SIZE   (sizeof(float4) * 2)
#define HIT_SIZE   (sizeof(int4))
#define NODE_SIZE  (sizeof(float4) * 4)
#define TRI_SIZE   (sizeof(float4) * 3)
#define INDEX_SIZE (sizeof(int))

    // Copy data to device
    hipMemcpyHtoD((hipDeviceptr_t) rays_d,        rays_h,       numRays  * RAY_SIZE);
    hipMemcpyHtoD((hipDeviceptr_t) results_d,     results_h,    numRays  * HIT_SIZE);
    hipMemcpyHtoD((hipDeviceptr_t) nodesA_d,      nodesA_h,     numNodes * NODE_SIZE);
    hipMemcpyHtoD((hipDeviceptr_t) triA_d,        triA_h,       numTris  * TRI_SIZE);
    hipMemcpyHtoD((hipDeviceptr_t) triIndices_d,  triIndices_h, numTris  * INDEX_SIZE);
    
	trace<<<numBlocks, threadsPerBlock>>>(numRays, false, rays, results, nodesA, nodesB, nodesC, nodesD, trisA, trisB, trisC, triIndices);   
    hipDeviceSynchronize();

    // Copy data from device
    hipMemcpyDtoH(results_h, (hipDeviceptr_t) results_d, numRays * HIT_SIZE);
}

//------------------------------------------------------------------------
